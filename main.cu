#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "sha256.cuh"
#include <dirent.h>
#include <ctype.h>
#include <math.h>
#include <gmp.h>

#define ROUNDS 50000

#define THREADS 1000
#define BLOCKS 1000

struct BLOCK {
    BYTE  version[04];
    BYTE prevHash[32];
    BYTE mrklRoot[32];
    BYTE timeStmp[04];
    BYTE bitsDiff[04];

    BYTE diffTrgt[32];

    long int x;
};
void printHashD(BYTE *hash) {

    printf("Hash : ");
    for (int i=0; i<32; i++) {
        printf("%02X", hash[i]);
    }
    printf("\n");
}

void d_int_to_little_endian_hex(long int num, BYTE *hex) {
    for (int i = 0; i < sizeof(int); i++) {
        hex[i] = (num >> (i * 8)) & 0xFF;
    }
}

void d_reverse(BYTE* arr, int size) {
    int start = 0;
    int end = size - 1;

    while (start < end) {
        BYTE temp = arr[start];
        arr[start] = arr[end];
        arr[end] = temp;
        start++; end--;
    }
}

__device__ void printHash(BYTE *hash) {

    printf("Hash : ");
    for (int i=0; i<32; i++) {
        printf("%02X", hash[i]);
    }
    printf("\n");
}

__device__ void reverse(BYTE* arr, int size) {
    int start = 0;
    int end = size - 1;

    while (start < end) {
        BYTE temp = arr[start];
        arr[start] = arr[end];
        arr[end] = temp;
        start++; end--;
    }
}

__device__ void int_to_little_endian_hex(long int num, BYTE *hex) {
    for (int i = 0; i < sizeof(int); i++) {
        hex[i] = (num >> (i * 8)) & 0xFF;
    }
}

__device__ bool is_valid(BYTE* blockHash, BYTE* targetHash) {

    for(int i = 0; i < 32; i++) {
        if(blockHash[i] > targetHash[i]) {
            return false;
        } else if(blockHash[i] < targetHash[i]) {
            return true;
        }
    }
    return false;
}

__device__ bool foundHash;

__global__ void startMining(BLOCK* block) {
    if (foundHash)
        return;

    BYTE digest[32];
    long int random;
    for (long int round=0; round<ROUNDS; round++) {

        random = blockIdx.x * (THREADS * ROUNDS) + threadIdx.x * ROUNDS + (round + 1);
        
        BYTE nonce[sizeof(int)];
        int_to_little_endian_hex(random, nonce);

        SHA256_CTX ctx;
        sha256_init(&ctx);
        sha256_update(&ctx, block->version, sizeof(block->version));
        sha256_update(&ctx, block->prevHash, sizeof(block->prevHash));
        sha256_update(&ctx, block->mrklRoot, sizeof(block->mrklRoot));
        sha256_update(&ctx, block->timeStmp, sizeof(block->timeStmp));
        sha256_update(&ctx, block->bitsDiff, sizeof(block->bitsDiff));
        sha256_update(&ctx, nonce, sizeof(nonce));
        sha256_final(&ctx, digest);

        sha256_init(&ctx);
        sha256_update(&ctx, digest, 32);
        sha256_final(&ctx, digest);
        reverse(digest, 32);

        if (is_valid(digest, block->diffTrgt)) {
            printHash(digest);
            printHash(block->diffTrgt);
            block->x = random;
            foundHash = true;
        }
    }
}

void hexstring_to_bytearray(const char *hexstring, BYTE *bytearray) {
    size_t i;
    size_t str_len = strlen(hexstring);

    for (i = 0; i < (str_len / 2); i++) {
        sscanf(hexstring + 2*i, "%2hhx", &bytearray[i]);
    }
}

void get_target_str(long int bits, BYTE* target) {

    long int exp = 8*((bits >> 24)-3);
    long int mant = bits & 0xffffff;

    mpz_t result, multiplied;
    mpz_inits(result, multiplied, NULL);
    mpz_ui_pow_ui(result, 2, exp);

    mpz_set_ui(multiplied, mant);
    mpz_mul(multiplied, multiplied, result);
    size_t count;
    BYTE *hex_bytes = (BYTE *)mpz_export(NULL, &count, 1, sizeof(BYTE), 1, 0, multiplied);

    for (size_t i=0; i<(32-count); ++i) {
        target[i] = 0x00;
    }
    for (size_t i=(32-count); i<32; ++i) {
        target[i] = hex_bytes[i-(32-count)];
    }
    mpz_clears(result, multiplied, NULL);
    free(hex_bytes);
}

int main() {

    long int version = 616259584;
    char prevBlock[] = "0000000000000000000286859eb09e9d6ab0bd39f9d6b1d2bce0a842851aa5db";
    char mrklRoot[] = "50b430dd0d84c3ffb2c5fd6d6b84f88dfc0aa1a92eafd469c549f71ad0cde202";
    long int time = 1713102559;
    long int bits = 486089497;
    
    BLOCK *d_currentBlock;
    BLOCK h_currentBlock;

    hexstring_to_bytearray(prevBlock, h_currentBlock.prevHash);
    hexstring_to_bytearray(mrklRoot, h_currentBlock.mrklRoot);
    d_reverse(h_currentBlock.prevHash, 32); d_reverse(h_currentBlock.mrklRoot, 32);

    d_int_to_little_endian_hex(version, h_currentBlock.version);
    d_int_to_little_endian_hex(time, h_currentBlock.timeStmp);
    d_int_to_little_endian_hex(bits, h_currentBlock.bitsDiff);
    get_target_str(bits, h_currentBlock.diffTrgt);

    hipMalloc(&d_currentBlock, sizeof(BLOCK));
    hipMemcpy(d_currentBlock, &h_currentBlock, sizeof(BLOCK), hipMemcpyHostToDevice);

    startMining<<<BLOCKS, THREADS>>>(d_currentBlock);
    hipDeviceSynchronize();
    
    hipMemcpy(&h_currentBlock, d_currentBlock, sizeof(BLOCK), hipMemcpyDeviceToHost);
    hipFree(d_currentBlock);
    printf("x = %ld\n", h_currentBlock.x);

    return 0;
}
